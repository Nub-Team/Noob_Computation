#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
 
#include "definitions.h"

__global__ void multi(int *A, int *B, int *C){
	int cvalue = 0;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;


	if (row > DIM || col > DIM) return;

	for (int e = 0; e < DIM; ++e){
		cvalue += A[row*DIM + e] * B[e*DIM + col];
	}
	C[row*DIM + col] = cvalue;
}

void matrixmulti(int A[][DIM],int B[][DIM],int C[][DIM]){
	int *dev_a, *dev_b, *dev_c;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t err = hipMalloc((void**)&dev_a, ((DIM)*(DIM))*sizeof(int));
	
	printf("Cuda malloc A:%s \n", hipGetErrorString(err));
	err = hipMalloc((void**)&dev_b, ((DIM)*(DIM))*sizeof(int));
	
	printf("Cuda malloc B:%s \n", hipGetErrorString(err));
	err = hipMalloc((void**)&dev_c, ((DIM)*(DIM))*sizeof(int));
	printf("Cuda malloc C:%s \n", hipGetErrorString(err));

	err = hipMemcpy(dev_a, A, ((DIM*DIM))*sizeof(int), hipMemcpyHostToDevice);
	printf("Cuda memcpy to device A:%s \n", hipGetErrorString(err));
	err = hipMemcpy(dev_b, B, ((DIM*DIM))*sizeof(int), hipMemcpyHostToDevice);
	printf("Cuda memcpy to device B:%s \n", hipGetErrorString(err));

	dim3 dimBlock(BlockSize, BlockSize);
	dim3 dimGrid((DIM + dimBlock.x - 1) / dimBlock.x, (DIM + dimBlock.y - 1) / dimBlock.y);
	
	hipEventRecord(start);
	multi << < dimGrid, dimBlock >> >(dev_a, dev_b, dev_c);
	hipEventRecord(stop);

	err = hipMemcpy(C, dev_c, ((DIM*DIM))*sizeof(int), hipMemcpyDeviceToHost);
	printf("Cuda memcpy to HOST C:%s \n", hipGetErrorString(err));
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Elapsed time is %f ms\n", milliseconds);


	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
