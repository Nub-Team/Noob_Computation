#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <stdio.h>
#include "common.cuh"
#include "math.h"

const float TILE_SIZE = 1024;

__global__ void VectoraddKernel(float* Agpu, float* Bgpu, float* Cgpu, int size){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < size) {
		Cgpu[tid] = Agpu[tid] + Bgpu[tid];
	}
}

__global__ void VectorsubtractKernel(float* Agpu, float* Bgpu, float* Cgpu, int size){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < size) {
		Cgpu[tid] = Agpu[tid] - Bgpu[tid];
	}
}

__global__ void VectorscaleKernel(float* Agpu, float* Cgpu, float scaling, int size){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < size) {
		Cgpu[tid] = Agpu[tid] * scaling;
	}
}

bool addVectorGPU(float* M, float* N, float* P, int size) {
	int bytes = size * sizeof(float);
	float* Agpu, * Bgpu, * Cgpu;
	
	hipHostGetDevicePointer((void**)&Agpu, M, 0);
	hipHostGetDevicePointer((void**)&Bgpu, N, 0);
	hipHostGetDevicePointer((void**)&Cgpu, P, 0);
	
	dim3 dimBlock(TILE_SIZE);
	dim3 dimGrid((int)ceil((float)size / (float)TILE_SIZE));
	
	VectoraddKernel << <dimGrid, dimBlock >> > (Agpu, Bgpu, Cgpu, size);
	hipDeviceSynchronize();
	
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) {
		printf("Kernel failed: %s", hipGetErrorString(status));
		return false;
	}
	return true;
}

bool subtractVectorGPU(float* M, float* N, float* P, int size) {
	int bytes = size * sizeof(float);
	float* Agpu, * Bgpu, * Cgpu;
	
	hipHostGetDevicePointer((void**)&Agpu, M, 0);
	hipHostGetDevicePointer((void**)&Bgpu, N, 0);
	hipHostGetDevicePointer((void**)&Cgpu, P, 0);
	
	dim3 dimBlock(TILE_SIZE);
	dim3 dimGrid((int)ceil((float)size / (float)TILE_SIZE));
	
	VectorsubtractKernel << <dimGrid, dimBlock >> > (Agpu, Bgpu, Cgpu, size);
	hipDeviceSynchronize();
	//Return error if any 
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) {
		printf("Kernel failed: %s", hipGetErrorString(status));
		return false;
	}

	return true;
}

bool scaleVectorGPU(float* M, float* P, float scaling, int size) {
	int bytes = size * sizeof(float);
	float* Agpu, * Cgpu;
	
	hipHostGetDevicePointer((void**)&Agpu, M, 0);
	hipHostGetDevicePointer((void**)&Cgpu, P, 0);
	
	dim3 dimBlock(TILE_SIZE);
	dim3 dimGrid((int)ceil((float)size / (float)TILE_SIZE));
	
	VectorscaleKernel << <dimGrid, dimBlock >> > (Agpu, Cgpu, scaling, size);
	hipDeviceSynchronize();
	
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) {
		printf("Kernel failed: %s", hipGetErrorString(status));
		return false;
	}

	return true;
}